/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.cuh"

#include <raft/core/device_coo_matrix.hpp>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/core/resources.hpp>
#include <raft/sparse/matrix/preprocessing.cuh>
#include <raft/sparse/selection/knn.cuh>
#include <raft/util/cudart_utils.hpp>

#include <gtest/gtest.h>

#include <iostream>
#include <limits>
// #include <thrust/reduce.h>
// #include <thrust/fill.h>
// #include <thrust/functional.h>

namespace raft {
namespace sparse {

template <typename Type_f, typename Index_>
struct SparsePreprocessInputs {
  int n_rows;
  int n_cols;
  std::vector<Index_> rows_h;
  std::vector<Index_> columns_h;
  std::vector<Type_f> values_h;
};

template <typename Type_f, typename Index_>
class SparseTest : public ::testing::TestWithParam<SparsePreprocessInputs<Type_f, Index_>> {
 public:
  SparseTest()
    : params(::testing::TestWithParam<SparsePreprocessInputs<Type_f, Index_>>::GetParam()),
      stream(resource::get_cuda_stream(handle)),
      n_rows(params.n_rows),
      n_cols(params.n_cols),
      rows(params.rows_h.size(), stream),
      columns(params.columns_h.size(), stream),
      values(params.values_h.size(), stream),
      result(params.values_h.size(), stream)

  {
  }

 protected:
  void SetUp() override {}

  void Run(bool bm25_on)
  {
    int k               = 3;
    hipStream_t stream = raft::resource::get_cuda_stream(handle);
    auto rows           = raft::make_device_vector<int, int64_t>(handle, params.rows_h.size());
    auto columns        = raft::make_device_vector<int, int64_t>(handle, params.columns_h.size());
    auto values         = raft::make_device_vector<float, int64_t>(handle, params.values_h.size());
    auto result         = raft::make_device_vector<float, int64_t>(handle, params.values_h.size());

    raft::copy(rows.data_handle(), params.rows_h.data(), params.rows_h.size(), stream);
    raft::copy(columns.data_handle(), params.columns_h.data(), params.columns_h.size(), stream);

    raft::copy(values.data_handle(), params.values_h.data(), params.values_h.size(), stream);

    auto coo_struct_view = raft::make_device_coordinate_structure_view(
      rows.data_handle(), columns.data_handle(), params.n_rows, params.n_cols, int(values.size()));
    auto coo_matrix = raft::make_device_coo_matrix<float, int, int, int>(handle, coo_struct_view);
    raft::update_device<float>(
      coo_matrix.view().get_elements().data(), values.data_handle(), values.size(), stream);

    if (bm25_on) {
      sparse::matrix::encode_bm25<int, float>(handle, coo_matrix.view(), result.view());
    } else {
      sparse::matrix::encode_tfidf<int, float>(handle, coo_matrix.view(), result.view());
    }

    auto out_rows_coo =
      raft::make_device_vector<int, int64_t>(handle, coo_matrix.structure_view().get_n_rows() * k);
    auto out_dists_coo = raft::make_device_vector<float, int64_t>(
      handle, coo_matrix.structure_view().get_n_rows() * k);

    raft::sparse::neighbors::brute_force_knn<int, float>(coo_matrix,
                                                         coo_matrix,
                                                         out_rows_coo.data_handle(),
                                                         out_dists_coo.data_handle(),
                                                         k,
                                                         handle,
                                                         coo_matrix.structure_view().get_n_rows(),
                                                         coo_matrix.structure_view().get_n_rows(),
                                                         raft::distance::DistanceType::L1);

    RAFT_CUDA_TRY(hipStreamSynchronize(stream));

    ASSERT_TRUE(values.size() == result.size());
    //   raft::devArrMatch<Type_f>(, nnz, raft::Compare<Type_f>()));
  }

 protected:
  raft::resources handle;
  hipStream_t stream;

  SparsePreprocessInputs<Type_f, Index_> params;
  int n_rows, n_cols;
  rmm::device_uvector<Index_> rows, columns;
  rmm::device_uvector<Type_f> values, result;
  bool bm25;
};

using SparseTestFF = SparseTest<float, int>;
TEST_P(SparseTestFF, Result) { Run(false); }

using SparseTestFT = SparseTest<float, int>;
TEST_P(SparseTestFT, Result) { Run(true); }

const std::vector<SparsePreprocessInputs<float, int>> sparse_preprocess_inputs = {
  {9,                                                    // n_rows
   4,                                                    // n_cols
   {0, 3, 4, 5, 6, 7, 8, 9, 10, 11},                     // rows
   {0, 0, 1, 2, 2, 1, 1, 3, 2, 1},                       // cols
   {1.0, 2.0, 2.0, 1.0, 1.0, 3.0, 4.0, 2.0, 1.0, 3.0}},  // vals
};

INSTANTIATE_TEST_CASE_P(SparseTest, SparseTestFF, ::testing::ValuesIn(sparse_preprocess_inputs));
INSTANTIATE_TEST_CASE_P(SparseTest, SparseTestFT, ::testing::ValuesIn(sparse_preprocess_inputs));

}  // namespace sparse
}  // namespace raft